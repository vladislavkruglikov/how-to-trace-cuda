
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void add(int size, double* array, double* other, double* total) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (int g = i; g < size; g += blockDim.x * gridDim.x) {
        total[g] = array[g] + other[g];
    }

}


double difference(int size, double *total, double expected) {
    double maximum_difference = 0;
    for (int i = 0; i < size; ++i) {
        double expected_result = 3.0;
        double difference = abs(expected_result - total[i]);
        if (difference > maximum_difference) {
            maximum_difference = difference;
        }
    }

    return maximum_difference;
}


int main() {
    int size = 1 << 20;

    double *array = (double*) malloc(sizeof(double) * size);
    double *other = (double*) malloc(sizeof(double) * size);
    double *total = (double*) malloc(sizeof(double) * size);

    for (int i = 0; i < size; ++i) {
        array[i] = 1.0;
        other[i] = 2.0;
    }
    
    double *cuda_array;
    double *cuda_other;
    double *cuda_total;

    hipMalloc(&cuda_array, size * sizeof(double));
    hipMalloc(&cuda_other, size * sizeof(double));
    hipMalloc(&cuda_total, size * sizeof(double));

    hipMemcpy(cuda_array, array, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cuda_other, other, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cuda_total, total, size * sizeof(double), hipMemcpyHostToDevice);

    int threads_per_block_count = 1024;
    int blocks_count = (size + threads_per_block_count - 1) / threads_per_block_count;
    add<<<blocks_count, threads_per_block_count>>>(size, cuda_array, cuda_other, cuda_total);

    hipMemcpy(total, cuda_total, size * sizeof(double), hipMemcpyDeviceToHost);

    double maximum_difference = difference(size, total, 3);

    printf("Maximum difference is %f\n", maximum_difference);

    free(array);
    free(other);
    free(total);

    return 0;
}
