// One block one thread cuda kernel that basically
// runs a single thread such that the kernel must be
// just a loop over all arrays and such algorithm does
// not take advantage of cuda kernels



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void add(int size, double* array, double* other, double* total) {
    for (int k = 0; k < size; ++k) {
        total[k] = array[k] + other[k];
    }
}


double difference(int size, double *total, double expected) {
    double maximum_difference = 0;
    for (int i = 0; i < size; ++i) {
        double expected_result = 3.0;
        double difference = abs(expected_result - total[i]);
        if (difference > maximum_difference) {
            maximum_difference = difference;
        }
    }

    return maximum_difference;
}


int main() {
    int size = 1 << 20;

    double *array = (double*) malloc(sizeof(double) * size);
    double *other = (double*) malloc(sizeof(double) * size);
    double *total = (double*) malloc(sizeof(double) * size);

    for (int i = 0; i < size; ++i) {
        array[i] = 1.0;
        other[i] = 2.0;
    }
    
    double *cuda_array;
    double *cuda_other;
    double *cuda_total;

    hipMalloc(&cuda_array, size * sizeof(double));
    hipMalloc(&cuda_other, size * sizeof(double));
    hipMalloc(&cuda_total, size * sizeof(double));

    hipMemcpy(cuda_array, array, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cuda_other, other, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cuda_total, total, size * sizeof(double), hipMemcpyHostToDevice);

    add<<<1, 1>>>(size, cuda_array, cuda_other, cuda_total);

    hipMemcpy(total, cuda_total, size * sizeof(double), hipMemcpyDeviceToHost);

    double maximum_difference = difference(size, total, 3);

    printf("Maximum difference is %f\n", maximum_difference);

    free(array);
    free(other);
    free(total);

    return 0;
}
